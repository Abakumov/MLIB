
#include <hip/hip_runtime.h>
/**
 * @file pctdemo_processMandelbrotElement.cu
 * 
 * CUDA code to calculate the Mandelbrot Set on a GPU.
 * 
 * Copyright 2011 The MathWorks, Inc.
 */

/** Work out which piece of the global array this thread should operate on */ 
__device__ size_t calculateGlobalIndex() {
    // Which block are we?
    size_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    // Which thread are we within the block?
    size_t const localThreadIdx = threadIdx.x + blockDim.x * threadIdx.y;
    // How big is each block?
    size_t const threadsPerBlock = blockDim.x*blockDim.y;
    // Which thread are we overall?
    return localThreadIdx + globalBlockIndex*threadsPerBlock;

}

/** The actual Mandelbrot algorithm for a single location */ 
__device__ unsigned int doIterations( double const realPart0, 
                                      double const imagPart0, 
                                      unsigned int const maxIters ) {
    // Initialise: z = z0
    double realPart = realPart0;
    double imagPart = imagPart0;
    unsigned int count = 0;
    // Loop until escape
    while ( ( count <= maxIters )
            && ((realPart*realPart + imagPart*imagPart) <= 4.0) ) {
        ++count;
        // Update: z = z*z + z0;
        double const oldRealPart = realPart;
        realPart = realPart*realPart - imagPart*imagPart + realPart0;
        imagPart = 2.0*oldRealPart*imagPart + imagPart0;
    }
    return count;
}


/** Main entry point.
 * Works out where the current thread should read/write to global memory
 * and calls doIterations to do the actual work.
 */
__global__ void processMandelbrotElement( 
                      double * out, 
                      const double * x, 
                      const double * y,
                      const unsigned int maxIters, 
                      const unsigned int numel ) {
    // Work out which thread we are
    size_t const globalThreadIdx = calculateGlobalIndex();

    // If we're off the end, return now
    if (globalThreadIdx >= numel) {
        return;
    }
    
    // Get our X and Y coords
    double const realPart0 = x[globalThreadIdx];
    double const imagPart0 = y[globalThreadIdx];

    // Run the itearations on this location
    unsigned int const count = doIterations( realPart0, imagPart0, maxIters );
    out[globalThreadIdx] = log( double( count + 1 ) );
}
